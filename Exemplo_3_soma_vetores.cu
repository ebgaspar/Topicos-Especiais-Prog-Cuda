#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void addKernel( int* c , const int* a , const int* b )
{
	int i = threadIdx.x;
	c[ i ] = a[ i ] + b[ i ];
}

int main( )
{
	const int arraySize = 5;
	const int a[ arraySize ] = { 1, 2, 3, 4, 5 };
	const int b[ arraySize ] = { 10, 20, 30, 40, 50 };
	int c[ arraySize ] = { 0 };

	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Alocar espa�o na mem�ria do device
	cudaStatus = hipMalloc( ( void** ) &dev_c , arraySize * sizeof( int ) );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipMalloc failed!" );
		goto Error;
	}

	cudaStatus = hipMalloc( ( void** ) &dev_a , arraySize * sizeof( int ) );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipMalloc failed!" );
		goto Error;
	}

	cudaStatus = hipMalloc( ( void** ) &dev_b , arraySize * sizeof( int ) );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipMalloc failed!" );
		goto Error;
	}

	// Copia os vetores do host para a device
	cudaStatus = hipMemcpy( dev_a , a , arraySize * sizeof( int ) , hipMemcpyHostToDevice );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipMemcpy failed!" );
		goto Error;
	}

	cudaStatus = hipMemcpy( dev_b , b , arraySize * sizeof( int ) , hipMemcpyHostToDevice );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipMemcpy failed!" );
		goto Error;
	}

	// Executar o kernel
	addKernel << <1 , arraySize >> > ( dev_c , dev_a , dev_b );

	// Verificar se o kernel foi executado corretamente
	cudaStatus = hipGetLastError( );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "addKernel launch failed: %s\n" , hipGetErrorString( cudaStatus ) );
		goto Error;
	}

	// Espera o kernel terminar e retorna quaisquer erros encontrados durante a execu��o
	cudaStatus = hipDeviceSynchronize( );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipDeviceSynchronize returned error code %d after launching addKernel!\n" , cudaStatus );
		goto Error;
	}

	// Copia o resultado do device para a mem�ria do host.
	cudaStatus = hipMemcpy( c , dev_c , arraySize * sizeof( int ) , hipMemcpyDeviceToHost );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipMemcpy failed!" );
		goto Error;
	}

	printf( "{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n" , c[ 0 ] , c[ 1 ] , c[ 2 ] , c[ 3 ] , c[ 4 ] );

	// Limpa a mem�ria
Error:
	hipFree( dev_c );
	hipFree( dev_a );
	hipFree( dev_b );

	cudaStatus = hipDeviceReset( );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipDeviceReset failed!" );
		return 1;
	}

	return 0;
}

