#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void meu_kernel( void )
{
	printf( "Meu ID: %d\n" , threadIdx.x );
}

int main( )
{
	// Define a vari�vel de captura de erros
	hipError_t cudaStatus;

	// Informa o device a ser usado caso exista mais de 1
	cudaStatus = hipSetDevice( 0 );

	// Testa a fun��o hipSetDevice retornou erro
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipSetDevice falhou!  Existe dispositivo com suporte a CUDA instalado?" );
		fprintf( stderr , "\n\n%s" , hipGetErrorString( cudaStatus ) );
		goto Error;
	}

	fprintf( stdout , "Inicio\n" );

	meu_kernel << < 2 , 5 >> > ( );

	// Captura o �ltimo erro ocorrido
	cudaStatus = hipGetLastError( );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "meu_kernel falhou: %s\n" , hipGetErrorString( cudaStatus ) );
		goto Error;
	}

	// Sincroniza a execu��o do kernel com a CPU
	cudaStatus = hipDeviceSynchronize( );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipDeviceSynchronize retornou erro %d ap�s lan�amento do kernel!\n" , cudaStatus );
		goto Error;
	}
	fprintf( stdout , "Fim\n" );
Error:
	// Executa a limpeza GPU
	cudaStatus = hipDeviceReset( );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipDeviceReset falhou!" );
		return 1;
	}

	return 0;
}