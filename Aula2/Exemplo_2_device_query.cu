#include "hip/hip_runtime.h"


#include <stdio.h>
int main( int argc , char** argv )
{

	fprintf( stdout , "CUDA Device Query\n" );

	int deviceCount = 0;

	// Testa se existem dispositivos compat�veis com Cuda
	hipError_t cudaStatus = hipGetDeviceCount( &deviceCount );

	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "hipGetDeviceCount retornou c�digo: %d\n -> %s\n" , cudaStatus , hipGetErrorString( cudaStatus ) );
		exit( 1 );
	}

	// A fun��o retorna 0 caso n�o exista hardware que suporte cuda.
	if ( deviceCount == 0 )
	{
		fprintf( stdout , "N�o h� dispositivo compat�vel com CUDA\n" );
	}
	else
	{
		fprintf( stdout , "Detectado %d dispositivo(s) CUDA\n" , deviceCount );
	}

	int dev , driverVersion = 0 , runtimeVersion = 0;

	for ( dev = 0; dev < deviceCount; ++dev )
	{
		hipSetDevice( dev );
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties( &deviceProp , dev );

		fprintf( stdout , "\nDevice %d: \"%s\"\n" , dev , deviceProp.name );

		hipDriverGetVersion( &driverVersion );
		hipRuntimeGetVersion( &runtimeVersion );
		hipDriverGetVersion( &driverVersion );
		hipRuntimeGetVersion( &runtimeVersion );
		fprintf( stdout , "CUDA Driver Version / Runtime Version %d.%d / %d.%d\n" , driverVersion / 1000 , ( driverVersion % 100 ) / 10 , runtimeVersion / 1000 , ( runtimeVersion % 100 ) / 10 );
		fprintf( stdout , "CUDA Capability Major/Minor version number: %d.%d\n" , deviceProp.major , deviceProp.minor );
		fprintf( stdout , "QTD Multiprocessors: %d \n" , deviceProp.multiProcessorCount );
		fprintf( stdout , "Total constant memory:%zu bytes\n" , deviceProp.totalConstMem );
		fprintf( stdout , "Total shared memory per block:%zu bytes\n" , deviceProp.sharedMemPerBlock );
		fprintf( stdout , "Shared memory per multiprocessor:%zu bytes\n" , deviceProp.sharedMemPerMultiprocessor );
		fprintf( stdout , "Number of registers available per block:%d\n" , deviceProp.regsPerBlock );
	}

	return 0;
}